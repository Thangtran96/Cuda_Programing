#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
using namespace std;
ifstream fi("Data.txt");
ofstream fo("Ans.txt");

string DataInp[35];
vector<string> h_motif_find;
vector<string> h_motif_dis_H;

typedef pair <int, string> pis;
typedef vector < pis > vpis;
string dataMotif[35];
int l, d;

//device memory;
vector<string> d_motif_find;
vector<string> d_motif_dis_H;
__device__ vpis dataAns;

void ReadFileData()
{
	fi >> l >> d;
	fi.ignore();
	for (int i = 0; i < 20; ++i) {
		getline(fi, dataMotif[i]);
	}
}

void OutFileAns(string s[100], int x) {
	for (int i = 0; i < x; i++) fo << ">Sequence" << i << endl << s[i] << endl ;
}

void init_motif()
{
	string tmp;
	for (int i = 0; i < 20; ++i)
	{
		tmp = dataMotif[i];
		for (int j = 0; j < tmp.size() - l; ++j)
		{
			if (i == 0) {
				h_motif_find.push_back(tmp.substr(j, l) );
				h_motif_dis_H.push_back(tmp.substr(j, l));
			}
			else {
				h_motif_dis_H.push_back(tmp.substr(j, l));
			}
		}
	}
}

//int Haming(string s1, string s2) {
//	int res = 0;
//	for (int i = 0; i<s1.size(); ++i)
//	{
//		if (s1[i] != s2[i]) res++;
//	}
//	return res;
//}

__device__ int min_dis_ham;
__global__ string d_data[35];

__global__ void score_haming(string *s1, string *s2) {

}

__global__ void dis_haming(string *s, string *d_data) {
	int res_Sum = 0, temp_score = 999;
	int index = blockIdx.x * blockDim.x + threadIdx.x;

}

//__global__ void paternbracnching( const int *l, const int *d)
//{
//	int index = blockIdx.x * blockDim.x + threadIdx.x;
//	int size_str = d_data[1].size();
//	if (index < size_str - l)
//	{
//		string motif_f = d_data[1][index];
//
//	}
//}

int main()
{
	ReadFileData();
	/*OutFileAns(DataInp,20);
	vector<string> motifVector;
	string motif = DataInp[0].substr(0, 9);
	cout << "motif arbitrary " << motif << endl;
	for (int j = 0; j < 20; ++j) {
		for (int i = 1; i < DataInp[j].size() - l - 1; i++)
		{
			string motifTemp = DataInp[0].substr(i, l);
			int sroce = Haming(motif, motifTemp); fo << motifTemp << " " << sroce << endl;
			if (sroce < d) motifVector.push_back(motifTemp);
		}
		for (int it = 0; it < motifVector.size(); ++it)
		{
			cout << motifVector[it] << endl;
		}
	}*/
	/*for (int i = 1; i < DataInp[0].size() - l - 1; i++)
	{
		string motifTemp = DataInp[0].substr(i, l);
		int sroce = Haming(motif, motifTemp); fo << motifTemp << " " << sroce << endl;
		if (sroce < d) motifVector.push_back(motifTemp);
	}
	for (int it = 0; it < motifVector.size(); ++it)
	{
		cout << motifVector[it] << endl;
	}*/

	init_motif();

    return 0;
}

