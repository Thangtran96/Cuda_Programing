#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
using namespace std;
ifstream fi("Data.txt");
ofstream fo("Ans.txt");

string DataInp[35];

typedef pair <int, string> pis;
typedef vector < pis > vpis;
string dataMotif[35];
int l, d;
__device__ vpis dataAns;

void ReadFileData()
{
	int temp; string s;
	fi >> temp; //cout << temp;
	for (int i = 1; i < 21; ++i) {
		//if (i % 2 == 0) continue;
		fi >> s;
		DataInp[i] = s;
	}
}

void OutFileAns(string s[100], int x) {
	for (int i = 0; i < x; i++) fo << ">Sequence" << i << endl << s[i] << endl ;
}

//int Haming(string s1, string s2) {
//	int res = 0;
//	for (int i = 0; i<s1.size(); ++i)
//	{
//		if (s1[i] != s2[i]) res++;
//	}
//	return res;
//}

__device__ int min_dis_ham;
__global__ string d_data[35];

__global__ void dis_haming(string *s) {
	int res_Sum = 0, temp_score = 999;
	for (int i = 1; i<=20; ++i)
	{
		string s1 = d_data[i];
		temp_score = 999;
		for (int j = 0; j < s1.size() - l + 1; ++j)
		{
			string temp_str = s1.substr(j, l);
			int score_lmer = 0;
			for (int k = 0; i < temp_str.size(); ++i) {
				if (s[i] != temp_str[i]) score_lmer++;
			}
			temp_score = score_lmer
		}
		res_Sum += temp_score;
	}
	min_dis_ham = res_Sum;
}

__global__ void paternbracnching( const int *l, const int *d)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int size_str = d_data[1].size();
	if (index < size_str - l)
	{
		string motif_f = d_data[1][index];

	}
}

int main()
{
	ReadFileData();
	/*OutFileAns(DataInp,20);
	vector<string> motifVector;
	string motif = DataInp[0].substr(0, 9);
	cout << "motif arbitrary " << motif << endl;
	for (int j = 0; j < 20; ++j) {
		for (int i = 1; i < DataInp[j].size() - l - 1; i++)
		{
			string motifTemp = DataInp[0].substr(i, l);
			int sroce = Haming(motif, motifTemp); fo << motifTemp << " " << sroce << endl;
			if (sroce < d) motifVector.push_back(motifTemp);
		}
		for (int it = 0; it < motifVector.size(); ++it)
		{
			cout << motifVector[it] << endl;
		}
	}*/
	/*for (int i = 1; i < DataInp[0].size() - l - 1; i++)
	{
		string motifTemp = DataInp[0].substr(i, l);
		int sroce = Haming(motif, motifTemp); fo << motifTemp << " " << sroce << endl;
		if (sroce < d) motifVector.push_back(motifTemp);
	}
	for (int it = 0; it < motifVector.size(); ++it)
	{
		cout << motifVector[it] << endl;
	}*/
    return 0;
}

