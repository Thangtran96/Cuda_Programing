
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
using namespace std;
ifstream fi("Data.txt");
ofstream fo("Ans.txt");

string DataInp[100];
string DataAns[100];

void ReadFileData()
{
	int temp; string s;
	fi >> temp; //cout << temp;
	for (int i = 0; i < temp; ++i) {
		//if (i % 2 == 0) continue;
		fi >> s;
		DataInp[i] = s;
	}
}

void OutFileAns(string s[100], int x) {
	for (int i = 0; i < x; i++) fo << ">Sequence" << i << endl << s[i] << endl ;
}

int Haming(string s1, string s2) {
	int res = 0;
	for (int i = 0; i<s1.size(); ++i)
	{
		if (s1[i] != s2[i]) res++;
	}
	return res;
}

int main()
{
	ReadFileData();
	OutFileAns(DataInp,20);
	int l = 9, d = 2;
	vector<string> motifVector;
	string motif = DataInp[0].substr(0, 9);
	cout << "motif arbitrary " << motif << endl;
	for (int j = 0; j < 20; ++j) {
		for (int i = 1; i < DataInp[j].size() - l - 1; i++)
		{
			string motifTemp = DataInp[0].substr(i, l);
			int sroce = Haming(motif, motifTemp); fo << motifTemp << " " << sroce << endl;
			if (sroce < d) motifVector.push_back(motifTemp);
		}
		for (int it = 0; it < motifVector.size(); ++it)
		{
			cout << motifVector[it] << endl;
		}
	}
	/*for (int i = 1; i < DataInp[0].size() - l - 1; i++)
	{
		string motifTemp = DataInp[0].substr(i, l);
		int sroce = Haming(motif, motifTemp); fo << motifTemp << " " << sroce << endl;
		if (sroce < d) motifVector.push_back(motifTemp);
	}
	for (int it = 0; it < motifVector.size(); ++it)
	{
		cout << motifVector[it] << endl;
	}*/
    return 0;
}

