#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
using namespace std;
ifstream fi("Data.txt");
ofstream fo("Ans.txt");
string s1, s2;
int ans = 0;
//
//__device__ int* d_arrayA;
//int* h_arrayA;

__global__ void cube(float * d_out, float * d_in) {
	// Todo: Fill in this function
	int idx = threadIdx.x;
	float f = d_in[idx];
	d_out[idx] = f * f * f;
	//d_arrayA[idx] = f * f * f;
}

int main(void) {
	//int a, b, c;
	//int *d_a, *d_b, *d_c;
	//int size = sizeof(int);

	////allocate space for device copies
	//hipMalloc((void **)&d_a, size);

	/*getline(fi, s1);
	getline(fi, s2);
	cout << "input:\n";
	cout << s1 << "\n" << s2;
	cout << "\nprocsess .....";
	for (int i = 0; i<s1.size(); ++i)
	{
		if (s1[i] != s2[i]) ans++;
	}
	cout << "\nans: " << ans << "\n";*/


	const int ARRAY_SIZE = 96;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

	// generate the input array on the host
	float h_in[ARRAY_SIZE];
	for (int i = 0; i < ARRAY_SIZE; i++) {
		h_in[i] = float(i);
	}
	float h_out[ARRAY_SIZE];

	// declare GPU memory pointers
	float * d_in;
	float * d_out;

	// allocate GPU memory
	hipMalloc((void**)&d_in, ARRAY_BYTES);
	hipMalloc((void**)&d_out, ARRAY_BYTES);

	// transfer the array to the GPU
	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

	// launch the kernel
	cube << <1, ARRAY_SIZE >> >(d_out, d_in);

	// copy back the result array to the CPU
	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	// print out the resulting array
	for (int i = 0; i < ARRAY_SIZE; i++) {
		printf("%f", h_out[i]);
		printf(((i % 4) != 3) ? "\t" : "\n");
	}

	/*hipMalloc(&h_arrayA, 12000 * sizeof(int));
	if ( hipMemcpyToSymbol(HIP_SYMBOL(d_arrayA), &h_arrayA, sizeof(h_arrayA) ) == hipSuccess ) {
		cout << " copy du lieu thanh cong" << endl;
	}
	if (hipMemcpyFromSymbol(&h_arrayA, HIP_SYMBOL(d_arrayA), sizeof(h_arrayA)) == hipSuccess) {
		cout << " copy du lieu thanh cong" << endl;
	}
	cout << &h_arrayA << endl;
	hipFree(d_arrayA);*/
	hipFree(d_in);
	hipFree(d_out);

	return 0;
}