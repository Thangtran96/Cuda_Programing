
#include "hip/hip_runtime.h"

#include<iostream>
#include <stdio.h>


using namespace std;

struct d_Motif
{
	int dis;
	int *loc;
};

struct h_Motif
{
	int dis;
	int loc[40];
};
__global__ void forKernel(int *a,int *b, d_Motif *c)
{
	//printf("a: %d | b: %d \n", a[0], b[0]);
	//10000005
	for (int i = 0; i < 105; ++i) {
		a[0] += b[0];
	}
	/*c[0].dis = 100;
	for (int i = 0; i < 20; ++i) {
		c[0].loc[i] = i;
	}*/
}

const size_t Array_byte = 3 * sizeof(int);

int main()
{
	int h_a[3] = { 0,1,2 }, h_b[3] = { 1,2,3 };
	cout << h_a[0] << " " << h_b[0] << endl;
	int *d_a, *d_b;
	if (hipMalloc(&d_a, Array_byte) != hipSuccess) {
		cout << "error allocating memory!" << endl;
		return 0;
	}
	if (hipMalloc(&d_b, Array_byte)!= hipSuccess) {
		cout << "error allocating memory!" << endl;
		hipFree(d_a);
		return 0;
	}
	d_Motif *h_c;
	size_t sizeMotif = 2 * 41 * sizeof(int);
	d_Motif *d_c;
	if (hipMalloc(&d_c, sizeMotif) != hipSuccess) {
		cout << "Error allocating memory struct!" << endl;
		hipFree(d_a);
		hipFree(d_b);
		return 0;
	}
	if ( hipMemcpy(d_a,h_a,Array_byte,hipMemcpyHostToDevice) != hipSuccess) {
		cout << "error copying memory!3"<<endl;
		hipFree(d_a);
		hipFree(d_b);
		hipFree(d_c);
		return 0;
	}
	if ( hipMemcpy(d_b, h_b, Array_byte, hipMemcpyHostToDevice) != hipSuccess) {
		cout << "error copying memory!2" << endl;
		hipFree(d_a);
		hipFree(d_b);
		hipFree(d_c);
		return 0;
	}
	if (hipMemcpy(d_c,h_c,sizeMotif,hipMemcpyHostToDevice) != hipSuccess) {
		cout << "error copying memory! struct!" << endl;
		hipFree(d_a);
		hipFree(d_b);
		hipFree(d_c);
		return 0;
	}

	forKernel<<<1, 1>>>(d_a, d_b, d_c);


	if (hipMemcpy(h_a,d_a,Array_byte,hipMemcpyDeviceToHost) != hipSuccess) {
		cout << "error copying memory!1" << endl;
		hipFree(d_a);
		hipFree(d_b);
		hipFree(d_c);
		return 0;
	}
	if (hipMemcpy(h_c, d_c, sizeMotif, hipMemcpyDeviceToHost) != hipSuccess) {
		cout << "Error copying memory!" << endl;
		hipFree(d_a);
		hipFree(d_b);
		hipFree(d_c);
		return 0;
	}
	cout << "adding 1 to 0 :" << h_a[0] << endl;
	//cout << "struct: " << *(h_c[0].dis) << " loc: " << *(h_c[0]->loc);
	int temp = h_c->dis;
	cout << "struct: " << temp ;

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	hipDeviceReset();
    return 0;
}

