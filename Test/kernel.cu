#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <string.h>
#include <vector>
#include <utility>
using namespace std;
const int ARRAY_SIZE = 12005;
const int ARRAY_BYTES_INT = ARRAY_SIZE * sizeof(int);

typedef struct {
	int dis;// dis_Haming
	int* motif;//str_motif[40]
}ans_motif;

//Host memory
int l, d;
int h_dataMotif[ARRAY_SIZE];

//device memory
__device__ ans_motif test;

//nhap tu file va chuyen sang kieu int
void File_Input()
{
	l = 9; d = 2;
	char temp[ARRAY_SIZE];
	FILE * pFile;
	pFile = fopen("datacu.txt", "r");
	if (pFile == NULL)
		perror("Error opening file");
	else
	{
		if (fgets(temp, ARRAY_SIZE, pFile) != NULL)
			cout << "nhap du lieu thanh cong!\n";
		fclose(pFile);
	}

	for (int i = 0; i < strlen(temp); ++i) {
		//A=0 C=1 G=2 T=3
		switch (temp[i])
		{
		case 'A': { h_dataMotif[i] = 0; break; }
		case 'C': { h_dataMotif[i] = 1; break; }
		case 'G': { h_dataMotif[i] = 2; break; }
		case 'T': { h_dataMotif[i] = 3; break; }
		default: cout << "error chuyen sang int";
			break;
		}
	}
}

//code dis_haming gpu
__device__ int dis_haming(const int* d_datainp, const int* s1, const int l) {
	//printf("\n dis_ham %d %d", s1, l);
	int ans = 0;
	int temp, tempRow;
	for (int i = 0; i < 20; ++i)
	{
		tempRow = 999;
		for (int j = i * 600; j < (i + 1) * 600 - l; ++j)
		{
			temp = 0;
			for (int k = 0; k < l; k++) {
				if (s1[k] != d_datainp[k + j]) temp++;
			}
			if (temp < tempRow) tempRow = temp;
		}
		ans += tempRow;
	}
	//printf("device code %d", ans);
	return ans;
}

//code best nay
//__device__ int bestNeighbor(const int* d_datainp, int* s1, const int l) {
//	int temp_val;
//	int temp_dis;
//	int best_dis = 99999;
//	int motif[40];
//	int ans = 0;
//	printf("\nbestNeighbor\n");
//	for (int i = 0; i < l; ++i) {
//		//trg hop 0
//		if (s1[i] != 0) {
//			temp_val = s1[i];
//			s1[i] = 0;
//			temp_dis = dis_haming(d_datainp, s1, l);
//			s1[i] = temp_val;
//			//lay best neighbor
//			if (temp_dis < best_dis)
//			{
//				best_dis = temp_dis;
//				for (int j = 0; j < l; ++j) {
//					motif[j] = s1[j];
//				}
//			}
//		}
//		//trg hop 1
//		if (s1[i] != 1) {
//			temp_val = s1[i];
//			s1[i] = 1;
//			temp_dis = dis_haming(d_datainp, s1, l);
//			s1[i] = temp_val;
//			//lay best neighbor
//			if (temp_dis < best_dis)
//			{
//				best_dis = temp_dis;
//				for (int j = 0; j < l; ++j) {
//					motif[j] = s1[j];
//				}
//			}
//		}
//		//trg hop 2
//		if (s1[i] != 2) {
//			temp_val = s1[i];
//			s1[i] = 2;
//			temp_dis = dis_haming(d_datainp, s1, l);
//			s1[i] = temp_val;
//			//lay best neighbor
//			if (temp_dis < best_dis)
//			{
//				best_dis = temp_dis;
//				for (int j = 0; j < l; ++j) {
//					motif[j] = s1[j];
//				}
//			}
//		}
//		//trg hop 3
//		if (s1[i] != 3) {
//			temp_val = s1[i];
//			s1[i] = 3;
//			temp_dis = dis_haming(d_datainp, s1, l);
//			s1[i] = temp_val;
//			//lay best neighbor
//			if (temp_dis < best_dis)
//			{
//				best_dis = temp_dis;
//				for (int j = 0; j < l; ++j) {
//					motif[j] = s1[j];
//				}
//			}
//		}
//	}
//	// chuan bi trc khi tra kq
//	int k = 0;
//	for (int i = 0; i < l; ++i) {
//		ans = ans ^ (motif[i] << k);
//		k += 2;
//	}
//	return ans;
//	printf("ans %d \n", ans);
//}

//code ham chinh goi
__global__ void patternBarching(const int* d_datainp, const int l, const int d) {
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < 600 - l) {
		//khai bao bien
		int motif_temp[40];
		int temp_val;
		int temp_dis;
		int best_dis = 99999;
		int motif_bN[40];
		int score_motif;
		
		//lay chuoi can duyet
		for (int i = 0; i < l; ++i) {
			motif_temp[i] = d_datainp[i + index];
		}
		score_motif = dis_haming(d_datainp, motif_temp, l);

		for (int k = 0; k < d; ++k) {
			//kiem tra chuoi tot
			if (best_dis < score_motif) {
				score_motif = best_dis;
				for (int i = 0; i < l; ++i) {
					motif_temp[i] = motif_bN[i];
				}
			}
			//ham bestNeighbor
			printf("\nbestNeighbor\n");
			for (int i = 0; i < l; ++i) {
				//trg hop 0
				if (motif_temp[i] != 0) {
					temp_val = motif_temp[i];
					motif_temp[i] = 0;
					temp_dis = dis_haming(d_datainp, motif_temp, l);
					//lay best neighbor
					if (temp_dis < best_dis)
					{
						best_dis = temp_dis;
						for (int j = 0; j < l; ++j) {
							motif_bN[j] = motif_temp[j];
						}
					}
					motif_temp[i] = temp_val;
				}
				//trg hop 1
				if (motif_temp[i] != 1) {
					temp_val = motif_temp[i];
					motif_temp[i] = 1;
					temp_dis = dis_haming(d_datainp, motif_temp, l);
					//lay best neighbor
					if (temp_dis < best_dis)
					{
						best_dis = temp_dis;
						for (int j = 0; j < l; ++j) {
							motif_bN[j] = motif_temp[j];
						}
					}
					motif_temp[i] = temp_val;
				}
				//trg hop 2
				if (motif_temp[i] != 2) {
					temp_val = motif_temp[i];
					motif_temp[i] = 2;
					temp_dis = dis_haming(d_datainp, motif_temp, l);
					//lay best neighbor
					if (temp_dis < best_dis)
					{
						best_dis = temp_dis;
						for (int j = 0; j < l; ++j) {
							motif_bN[j] = motif_temp[j];
						}
					}
					motif_temp[i] = temp_val;
				}
				//trg hop 3
				if (motif_temp[i] != 3) {
					temp_val = motif_temp[i];
					motif_temp[i] = 3;
					temp_dis = dis_haming(d_datainp, motif_temp, l);
					//lay best neighbor
					if (temp_dis < best_dis)
					{
						best_dis = temp_dis;
						for (int j = 0; j < l; ++j) {
							motif_bN[j] = motif_temp[j];
						}
					}
					motif_temp[i] = temp_val;
				}
			}
			// END ham bestNeighbor
		}

		

		/*for (int k = 0; k <= d; k++)
		{
			i_temp = dis_hamming(a);
			fo << i_temp << " " << bestScore << endl;
			if (i_temp < bestScore)
			{
				cout << "Change ";
				cout << a << " " << i_temp << " || ";
				f = a;
				bestScore = i_temp;
			}
			a = bestNeighbor(a);
		}*/
	}
}


int main() {
	File_Input();
	cout << h_dataMotif[1];
	//khai bao gpu
	int* d_datainp;
	if (hipMalloc(&d_datainp, ARRAY_BYTES_INT) == hipSuccess)
		cout << "\n Khai bao thanh cong\n";
	if (hipMemcpy(d_datainp, h_dataMotif, ARRAY_BYTES_INT, hipMemcpyHostToDevice) == hipSuccess)
		cout << "\n copy thanh cong\n";
	ans_motif* d_motif;

	//chay gpu
	patternBarching << < 1, 1 >> > (d_datainp, l, d);
	//hipMemcpy(h_datainp, d_datainp, ARRAY_BYTES, hipMemcpyDeviceToHost);
	hipFree(d_datainp);

	return 0;
}

