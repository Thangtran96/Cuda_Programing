#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#define N 5

__device__ float add_func(float x, float y)
{
	return x + y;
}

__device__ float mul_func(float x, float y)
{
	return x * y;
}

__device__ float div_func(float x, float y)
{
		return x / y;

}

typedef float(*op_func) (float, float);

__device__ op_func func[3] = { add_func, mul_func, div_func };
__device__ char* op_name[3] = { "add", "mul", "div" };


__device__ void op_array(const float *a, const float *b, float *res, int op, int n)
{
	for (int i = 0; i < N; i++) {
		res[i] = func[op](a[i], b[i]);
	}	
}

__global__ void kernel(void)
{

	float x[N];
	float y[N];
	float res[N];

	for (int i = 0; i < N; i++) {
		x[i] = (float)(10 + i);
	}
	for (int i = 0; i < N; i++) {
		y[i] = (float)(100 + i);
	}

	for (int op = 0; op < 3; op++) {
		printf("\nop=%s\n", op_name[op]);
		op_array(x, y, res, op, N);
		for (int i = 0; i < N; i++) {
			printf("res = % 16.9e\n", res[i]);
		}
	}
}



int main(void)
{
	kernel << <1, 1 >> > ();
	hipDeviceSynchronize();
	return EXIT_SUCCESS;
}