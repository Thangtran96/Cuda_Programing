
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <string.h>
#include <vector>
#include <utility>
using namespace std;
const int ARRAY_SIZE = 12005;
const int ARRAY_BYTES_INT = ARRAY_SIZE * sizeof(int);
const int ARRAY_SIZE_MOTIF = 601;
const int ARRAY_BYTES_MOTIF = ARRAY_SIZE_MOTIF * ( sizeof(int) + ( 40 * sizeof(int) ) );

typedef struct {
	int dis;// dis_Haming
	int* motif;//str_motif[40]
}MOTIF;

//Host memory
int l, d;
int h_dataMotif[12005];
int* d_datainp;
MOTIF h_motif[602];
MOTIF* d_motif;

//device memory

//nhap tu file va chuyen sang kieu int
void File_Input()
{
	l = 9; d = 2;
	char temp[ARRAY_SIZE];
	FILE * pFile;
	pFile = fopen("datacu.txt", "r");
	if (pFile == NULL)
		perror("Error opening file");
	else
	{
		if (fgets(temp, ARRAY_SIZE, pFile) != NULL)
			cout << "nhap du lieu thanh cong!\n";
		fclose(pFile);
	}

	for (int i = 0; i < strlen(temp); ++i) {
		//A=0 C=1 G=2 T=3
		switch (temp[i])
		{
		case 'A': { h_dataMotif[i] = 0; break; }
		case 'C': { h_dataMotif[i] = 1; break; }
		case 'G': { h_dataMotif[i] = 2; break; }
		case 'T': { h_dataMotif[i] = 3; break; }
		default: cout << "error chuyen sang int";
			break;
		}
	}
}

//code dis_haming gpu
__device__ int dis_haming(const int* d_datainp, const int* s1, const int l) {
	//printf("\n dis_ham %d %d", s1, l);
	int ans = 0;
	int temp, tempRow;
	for (int i = 0; i < 20; ++i)
	{
		tempRow = 999;
		for (int j = i * 600; j < (i + 1) * 600 - l; ++j)
		{
			temp = 0;
			for (int k = 0; k < l; k++) {
				if (s1[k] != d_datainp[k + j]) temp++;
			}
			if (temp < tempRow) tempRow = temp;
		}
		ans += tempRow;
	}
	//printf("device code %d", ans);
	return ans;
}

//code best nay
//__device__ int bestNeighbor(const int* d_datainp, int* s1, const int l) {
//	int temp_val;
//	int temp_dis;
//	int best_dis = 99999;
//	int motif[40];
//	int ans = 0;
//	printf("\nbestNeighbor\n");
//	for (int i = 0; i < l; ++i) {
//		//trg hop 0
//		if (s1[i] != 0) {
//			temp_val = s1[i];
//			s1[i] = 0;
//			temp_dis = dis_haming(d_datainp, s1, l);
//			s1[i] = temp_val;
//			//lay best neighbor
//			if (temp_dis < best_dis)
//			{
//				best_dis = temp_dis;
//				for (int j = 0; j < l; ++j) {
//					motif[j] = s1[j];
//				}
//			}
//		}
//		//trg hop 1
//		if (s1[i] != 1) {
//			temp_val = s1[i];
//			s1[i] = 1;
//			temp_dis = dis_haming(d_datainp, s1, l);
//			s1[i] = temp_val;
//			//lay best neighbor
//			if (temp_dis < best_dis)
//			{
//				best_dis = temp_dis;
//				for (int j = 0; j < l; ++j) {
//					motif[j] = s1[j];
//				}
//			}
//		}
//		//trg hop 2
//		if (s1[i] != 2) {
//			temp_val = s1[i];
//			s1[i] = 2;
//			temp_dis = dis_haming(d_datainp, s1, l);
//			s1[i] = temp_val;
//			//lay best neighbor
//			if (temp_dis < best_dis)
//			{
//				best_dis = temp_dis;
//				for (int j = 0; j < l; ++j) {
//					motif[j] = s1[j];
//				}
//			}
//		}
//		//trg hop 3
//		if (s1[i] != 3) {
//			temp_val = s1[i];
//			s1[i] = 3;
//			temp_dis = dis_haming(d_datainp, s1, l);
//			s1[i] = temp_val;
//			//lay best neighbor
//			if (temp_dis < best_dis)
//			{
//				best_dis = temp_dis;
//				for (int j = 0; j < l; ++j) {
//					motif[j] = s1[j];
//				}
//			}
//		}
//	}
//	// chuan bi trc khi tra kq
//	int k = 0;
//	for (int i = 0; i < l; ++i) {
//		ans = ans ^ (motif[i] << k);
//		k += 2;
//	}
//	return ans;
//	printf("ans %d \n", ans);
//}

//code ham chinh goi
__global__ void patternBarching(const int* d_datainp, const int l, const int d, MOTIF* d_motif) {
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < 600 - l) {
		//khai bao bien
		int motif_temp[40];
		int temp_val;
		int temp_dis;
		int best_dis = 99999;
		int motif_bN[40];
		int score_motif;
		
		//lay chuoi can duyet
		for (int i = 0; i < l; ++i) {
			motif_temp[i] = d_datainp[i + index];
			motif_bN[i] = motif_temp[i];
		}
		printf("\n 1 \n");
		score_motif = dis_haming(d_datainp, motif_temp, l);
		//ham dis_hamming
		//int ans = 0;
		//int temp, tempRow;
		//for (int i = 0; i < 20; ++i)
		//{
		//	tempRow = 999;
		//	for (int j = i * 600; j < (i + 1) * 600 - l; ++j)
		//	{
		//		temp = 0;
		//		for (int k = 0; k < l; k++) {
		//			if (s1[k] != d_datainp[k + j]) temp++;
		//		}
		//		if (temp < tempRow) tempRow = temp;
		//	}
		//	ans += tempRow;
		//}
		//printf("device code %d", ans);

		//chay ham patternBarching
		for (int k = 0; k < d; ++k) {
			//kiem tra chuoi tot
			printf("\n 2 \n");
			if (best_dis < score_motif) {
				score_motif = best_dis;
				for (int i = 0; i < l; ++i) {
					motif_temp[i] = motif_bN[i];
				}
			}
			//ham bestNeighbor
			printf("\nbestNeighbor\n");
			for (int i = 0; i < l; ++i) {
				printf("\n 3 \n");
				//trg hop 0
				if (motif_temp[i] != 0) {
					temp_val = motif_temp[i];
					motif_temp[i] = 0;
					temp_dis = dis_haming(d_datainp, motif_temp, l);
					//lay best neighbor
					if (temp_dis < best_dis)
					{
						best_dis = temp_dis;
						for (int j = 0; j < l; ++j) {
							motif_bN[j] = motif_temp[j];
						}
					}
					motif_temp[i] = temp_val;
				}
				//trg hop 1
				if (motif_temp[i] != 1) {
					temp_val = motif_temp[i];
					motif_temp[i] = 1;
					temp_dis = dis_haming(d_datainp, motif_temp, l);
					//lay best neighbor
					if (temp_dis < best_dis)
					{
						best_dis = temp_dis;
						for (int j = 0; j < l; ++j) {
							motif_bN[j] = motif_temp[j];
						}
					}
					motif_temp[i] = temp_val;
				}
				//trg hop 2
				if (motif_temp[i] != 2) {
					temp_val = motif_temp[i];
					motif_temp[i] = 2;
					temp_dis = dis_haming(d_datainp, motif_temp, l);
					//lay best neighbor
					if (temp_dis < best_dis)
					{
						best_dis = temp_dis;
						for (int j = 0; j < l; ++j) {
							motif_bN[j] = motif_temp[j];
						}
					}
					motif_temp[i] = temp_val;
				}
				//trg hop 3
				if (motif_temp[i] != 3) {
					temp_val = motif_temp[i];
					motif_temp[i] = 3;
					temp_dis = dis_haming(d_datainp, motif_temp, l);
					//lay best neighbor
					if (temp_dis < best_dis)
					{
						best_dis = temp_dis;
						for (int j = 0; j < l; ++j) {
							motif_bN[j] = motif_temp[j];
						}
					}
					motif_temp[i] = temp_val;
				}
			}
			// END ham bestNeighbor
		}
		printf("\n 4 \n");
		//du lieu tra lai
		printf("\n gan du lieu vao d_motif \n");
		d_motif[index].dis = score_motif;
		printf("\n d_ motif: dis = %d", d_motif[index].dis);
		for (int i = 0; i < l; ++i) {
			d_motif[index].motif[i] = motif_temp[i];
		}

		/*for (int k = 0; k <= d; k++)
		{
			i_temp = dis_hamming(a);
			fo << i_temp << " " << bestScore << endl;
			if (i_temp < bestScore)
			{
				cout << "Change ";
				cout << a << " " << i_temp << " || ";
				f = a;
				bestScore = i_temp;
			}
			a = bestNeighbor(a);
		}*/
	}
}


int main() {
	File_Input();
	//cout << h_dataMotif[1];	
	if (hipMalloc(&d_datainp, ARRAY_BYTES_INT) != hipSuccess)
		cout << "\n Khai bao ko thanh cong\n";
	if (hipMemcpy(d_datainp, h_dataMotif, ARRAY_BYTES_INT, hipMemcpyHostToDevice) != hipSuccess)
		cout << "\n copy ko thanh cong\n";
	
	if (hipMalloc(&d_motif, ARRAY_BYTES_MOTIF) != hipSuccess) {
		cout << "\n khai bao ko thanh cong d_motif\n";
	}
	//chay gpu
	patternBarching <<< 1, 1 >>> (d_datainp, l, d, d_motif);
	//cudaMemcpy(h_datainp, d_datainp, ARRAY_BYTES, cudaMemcpyDeviceToHost);
	if (hipMemcpy(h_motif, d_motif, ARRAY_BYTES_MOTIF, hipMemcpyDeviceToHost) != hipSuccess)
		cout << "\n ko copy dc data sang host" << endl;
	//cout << h_motif[0].dis << h_motif[0].motif[1];
	hipFree(d_datainp);
	hipFree(d_motif);

	return 0;
}

