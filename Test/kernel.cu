#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <string.h>
#include <vector>
#include <utility>
using namespace std;
const int ARRAY_SIZE = 12005;
const int ARRAY_BYTES_INT = ARRAY_SIZE * sizeof(int);

typedef struct {
	int dis;
	int* motif;
	int* pos;
}ans_motif;

//Host memory
int l, d;
int h_dataMotif[ARRAY_SIZE];

//device memory
__device__ ans_motif test;

//nhap tu file va chuyen sang kieu int
void File_Input()
{
	l = 9; d = 2;
	char temp[ARRAY_SIZE];
	FILE * pFile;
	pFile = fopen("datacu.txt", "r");
	if (pFile == NULL)
		perror("Error opening file");
	else
	{
		if (fgets(temp, ARRAY_SIZE, pFile) != NULL)
			cout << "nhap du lieu thanh cong!\n";
		fclose(pFile);
	}

	for (int i = 0; i < strlen(temp); ++i) {
		//A=0 C=1 G=2 T=3
		switch (temp[i])
		{
		case 'A': { h_dataMotif[i] = 0; break; }
		case 'C': { h_dataMotif[i] = 1; break; }
		case 'G': { h_dataMotif[i] = 2; break; }
		case 'T': { h_dataMotif[i] = 3; break; }
		default: cout << "error chuyen sang int";
			break;
		}
	}
}

//code dis_haming gpu
__device__ int dis_haming(const int* d_datainp, const int* s1, const int l) {
	//printf("\n dis_ham %d %d", s1, l);
	int ans = 0;
	int temp, tempRow;
	for (int i = 0; i < 20; ++i)
	{
		tempRow = 999;
		for (int j = i * 600; j < (i + 1) * 600 - l; ++j)
		{
			temp = 0;
			for (int k = 0; k < l; k++) {
				if (s1[k] != d_datainp[k + j]) temp++;
			}
			if (temp < tempRow) tempRow = temp;
		}
		ans += tempRow;
	}
	//printf("device code %d", ans);
	return ans;
}

//code best nay
__device__ int bestNeighbor(const int* d_datainp, int* s1, const int l) {
	int temp_val;
	int temp_dis;
	int best_dis=99999;
	int motif[40];
	int ans = 0;
	printf("\nbestNeighbor\n");
	for (int i = 0; i < l; ++i) {
		//trg hop 0
		if (s1[i] != 0) {
			temp_val = s1[i];
			s1[i] = 0;
			temp_dis = dis_haming(d_datainp, s1, l);
			s1[i] = temp_val;
			//lay best neighbor
			if (temp_dis < best_dis)
			{
				best_dis = temp_dis;
				for (int j = 0; j < l; ++j) {
					motif[j] = s1[j];
				}
			}
		}
		//trg hop 1
		if (s1[i] != 1) {
			temp_val = s1[i];
			s1[i] = 1;
			temp_dis = dis_haming(d_datainp, s1, l);
			s1[i] = temp_val;
			//lay best neighbor
			if (temp_dis < best_dis)
			{
				best_dis = temp_dis;
				for (int j = 0; j < l; ++j) {
					motif[j] = s1[j];
				}
			}
		}
		//trg hop 2
		if (s1[i] != 2) {
			temp_val = s1[i];
			s1[i] = 2;
			temp_dis = dis_haming(d_datainp, s1, l);
			s1[i] = temp_val;
			//lay best neighbor
			if (temp_dis < best_dis)
			{
				best_dis = temp_dis;
				for (int j = 0; j < l; ++j) {
					motif[j] = s1[j];
				}
			}
		}
		//trg hop 3
		if (s1[i] != 3) {
			temp_val = s1[i];
			s1[i] = 3;
			temp_dis = dis_haming(d_datainp, s1, l);
			s1[i] = temp_val;
			//lay best neighbor
			if (temp_dis < best_dis)
			{
				best_dis = temp_dis;
				for (int j = 0; j < l; ++j) {
					motif[j] = s1[j];
				}
			}
		}
	}
	// chuan bi trc khi tra kq
	int k = 0;
	for (int i = 0; i < l; ++i) {
		ans = ans ^ (motif[i] << k);
		k += 2;
	}
	return ans;
	printf("ans %d \n", ans);
}

//code ham chinh goi
__global__ void patternBarching(const int* d_datainp, const int l, const int d) {
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	int ans = 0;
	test.dis = 2;
	printf("\n thread chay %d %d %d", index, l, d);
	if (index < 600 - l) {
		int motif[40];
		for (int i = 0; i < l; ++i) {
			motif[i] = d_datainp[index + i];
		}
		printf("gia tri 1 trong %d \n", d_datainp[1]);
		ans = dis_haming(d_datainp, motif, l);
		printf("dis motif %d \n", ans);
		
		int ans_neighbor = bestNeighbor(d_datainp, motif, l);
		printf("best neighbor %d \n", ans_neighbor);
		int motifN[40];
		int x=0,k=0;
		for (int i = 0; i < l; ++i) {
			x = 3 << k;
			if (x & ans_neighbor == 0) { motifN[i] = 0; k += 2; printf("%d ", motifN[i]); continue; }
			if (x & ans_neighbor == 1) { motifN[i] = 1; k += 2; printf("%d ", motifN[i]); continue; }
			if (x & ans_neighbor == 2) { motifN[i] = 2; k += 2; printf("%d ", motifN[i]); continue; }
			if (x & ans_neighbor == 3) { motifN[i] = 3; k += 2; printf("%d ", motifN[i]); continue; }
		}
		printf("\n");
	}
}


int main() {
	File_Input();
	cout << h_dataMotif[1];
	int* d_datainp;
	if (hipMalloc(&d_datainp, ARRAY_BYTES_INT) == hipSuccess)
		cout << "\n Khai bao thanh cong\n";
	if (hipMemcpy(d_datainp, h_dataMotif, ARRAY_BYTES_INT, hipMemcpyHostToDevice) == hipSuccess)
		cout << "\n copy thanh cong\n";

	patternBarching << < 1, 1 >> > (d_datainp, l, d);
	//hipMemcpy(h_datainp, d_datainp, ARRAY_BYTES, hipMemcpyDeviceToHost);
	hipFree(d_datainp);

	return 0;
}

