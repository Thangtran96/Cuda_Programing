
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <string.h>
#include <vector>
#include <utility>
using namespace std;
const int ARRAY_SIZE = 12005;
const int ARRAY_BYTES_INT = ARRAY_SIZE * sizeof(int);

//Host memory
int l, d;
int h_dataMotif[ARRAY_SIZE];

//nhap tu file va chuyen sang kieu int
void File_Input()
{
	l = 9; d = 2;
	char temp[ARRAY_SIZE];
	FILE * pFile;
	pFile = fopen("datacu.txt", "r");
	if (pFile == NULL)
		perror("Error opening file");
	else
	{
		if (fgets(temp, ARRAY_SIZE, pFile) != NULL)
			cout << "nhap du lieu thanh cong!\n";
		fclose(pFile);
	}

	for (int i = 0; i < strlen(temp); ++i) {
		//A=0 C=1 G=2 T=3
		switch (temp[i])
		{
		case 'A':{ h_dataMotif[i] = 0; break;}
		case 'C': { h_dataMotif[i] = 1; break; }
		case 'G': { h_dataMotif[i] = 2; break; }
		case 'T': { h_dataMotif[i] = 3; break; }
		default: cout << "error chuyen sang int";
			break;
		}
	}
}

//code dis_haming gpu
__device__ int dis_haming(const int* d_datainp, const int* s1, const int l) {
	//printf("\n dis_ham %d %d", s1, l);
	int ans = 0;
	int temp, tempRow;
	for (int i = 0; i<20; ++i)
	{
		tempRow = 999;
		for (int j = i*600; j < (i+1)*600 - l; ++j)
		{
			temp = 0;
			for (int k = 0; k < l; k++) {
				if (s1[k] != d_datainp[k + j]) temp++;
			}
			if (temp < tempRow) tempRow = temp;
		}
		ans += tempRow;
	}
	//printf("device code %d", ans);
	return ans;
}

//code best nay
__device__ int* bestNeighbor(const int* d_datainp, const int* s1, const int l) {
	for (int i = 0; i < l; ++i) {
	}
}

//code ham chinh goi
__global__ void patternBarching(const int* d_datainp, const int l, const int d) {
	int index = blockDim.x * blockIdx.x + threadIdx.x ;
	int ans = 0;
	printf("\n thread chay %d %d %d", index, l, d);
	if (index < 600 - l) {
		int motif[40];
		for (int i = 0; i < l; ++i) {
			motif[i] = d_datainp[index + i];
		}
		printf("gia tri 1 trong %d \n", d_datainp[1]);
		ans = dis_haming(d_datainp, motif, l);
		printf("dis motif %d", ans);
	}
}


int main() {
	File_Input();
	cout << h_dataMotif[1];
	int* d_datainp;
	if (hipMalloc(&d_datainp, ARRAY_BYTES_INT) == hipSuccess)
		cout << "\n Khai bao thanh cong\n";
	if (hipMemcpy(d_datainp, h_dataMotif, ARRAY_BYTES_INT, hipMemcpyHostToDevice) == hipSuccess)
		cout << "\n copy thanh cong\n";

	patternBarching <<< 1, 1 >>> (d_datainp, l, d);
	
	//cudaMemcpy(h_datainp, d_datainp, ARRAY_BYTES, cudaMemcpyDeviceToHost);
	hipFree(d_datainp);

	return 0;
}

