#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <utility>
using namespace std;
ifstream fi("Data.txt");
ofstream fo("Ans.txt");

typedef pair <int, string> pis;
typedef vector < pis > vpis;

typedef pair <int, vector<int> > piv;
typedef pair < string, piv > pspiv;

//Host 
string h_DataInp[23];
int h_l, h_d;
vector<string> h_motif;

//Device
__device__ char d_datainp[23][650];
__device__ int d_l;
__device__ int d_d;

//chuoi khoi tao chuoi s0 de chay 20 luong, chay hamming tren 20 luong

__device__ int score_ham(char *s1, char *s2, )
{
	int res = 0;
	for (int i = 0; i < s1.size(); ++i) if (s1[i] != s2[i]) res++;
	return res;
}

__device__ int dis_Haming(string s, string *d_DataInp, int d_l)
{
	int res_Sum = 0, temp_score = 999;
	for (int i = 0; i < 20; ++i)
	{
		string s1 = d_DataInp[i];
		temp_score = 999;
		for (int j = 0; j < s1.size() - d_l + 1; ++j)
		{
			string temp_str = s1.substr(j, d_l);
			int score_s = score_ham(s, temp_str);
			if (temp_score > score_s)
			{
				temp_score = score_s;
			}
		}
		res_Sum += temp_score;
	}
	return res_Sum;
}

__device__ string bestNeighbor(string s, string *d_DataInp, int d_l)
{
	string temp_str = ""; //ATCG
	string ans;
	int diem = 999;
	int temp_dis;
	for (int i = 0; i < s.size(); ++i)
	{
		if (i == 0)
		{
			if (s[i] != 'A')
			{
				temp_str = 'A' + s.substr(1, s.size() - 1);
				temp_dis = dis_Haming(temp_str, d_DataInp, d_l);
				if (temp_dis <= diem) { 
					diem = temp_dis; 
					ans = temp_str;
				}
			}
			if (s[i] != 'T')
			{
				temp_str = 'T' + s.substr(1, s.size() - 1);
				temp_dis = dis_Haming(temp_str, d_DataInp, d_l);
				if (temp_dis <= diem) {
					diem = temp_dis;
					ans = temp_str;
				}
			}
			if (s[i] != 'G')
			{
				temp_str = 'G' + s.substr(1, s.size() - 1);
				temp_dis = dis_Haming(temp_str, d_DataInp, d_l);
				if (temp_dis <= diem) {
					diem = temp_dis;
					ans = temp_str;
				}
			}
			if (s[i] != 'C')
			{
				temp_str = 'C' + s.substr(1, s.size() - 1);
				temp_dis = dis_Haming(temp_str, d_DataInp, d_l);
				if (temp_dis <= diem) {
					diem = temp_dis;
					ans = temp_str;
				}
			}
			continue;
		}
		if (i == s.size() - 1)
		{
			if (s[i] != 'A')
			{
				temp_str = s.substr(0, s.size() - 1) + 'A';
				temp_dis = dis_Haming(temp_str, d_DataInp, d_l);
				if (temp_dis <= diem) {
					diem = temp_dis;
					ans = temp_str;
				}
			}
			if (s[i] != 'T')
			{
				temp_str = s.substr(0, s.size() - 1) + 'T';
				temp_dis = dis_Haming(temp_str, d_DataInp, d_l);
				if (temp_dis <= diem) {
					diem = temp_dis;
					ans = temp_str;
				}
			}
			if (s[i] != 'G')
			{
				temp_str = s.substr(0, s.size() - 1) + 'G';
				temp_dis = dis_Haming(temp_str, d_DataInp, d_l);
				if (temp_dis <= diem) {
					diem = temp_dis;
					ans = temp_str;
				}
			}
			if (s[i] != 'C')
			{
				temp_str = s.substr(0, s.size() - 1) + 'C';
				temp_dis = dis_Haming(temp_str, d_DataInp, d_l);
				if (temp_dis <= diem) {
					diem = temp_dis;
					ans = temp_str;
				}
			}
			break;
		}
		if (s[i] != 'A')
		{
			temp_str = s.substr(0, i) + 'A' + s.substr(i + 1, s.size() - i);
			temp_dis = dis_Haming(temp_str, d_DataInp, d_l);
			if (temp_dis <= diem) {
				diem = temp_dis;
				ans = temp_str;
			}
		}
		if (s[i] != 'T')
		{
			temp_str = s.substr(0, i) + 'T' + s.substr(i + 1, s.size() - i);
			temp_dis = dis_Haming(temp_str, d_DataInp, d_l);
			if (temp_dis <= diem) {
				diem = temp_dis;
				ans = temp_str;
			}
		}
		if (s[i] != 'G')
		{
			temp_str = s.substr(0, i) + 'G' + s.substr(i + 1, s.size() - i);
			temp_dis = dis_Haming(temp_str, d_DataInp, d_l);
			if (temp_dis <= diem) {
				diem = temp_dis;
				ans = temp_str;
			}
		}
		if (s[i] != 'C')
		{
			temp_str = s.substr(0, i) + 'C' + s.substr(i + 1, s.size() - i);
			temp_dis = dis_Haming(temp_str, d_DataInp, d_l);
			if (temp_dis <= diem) {
				diem = temp_dis;
				ans = temp_str;
			}
		}
	}
	return ans;
}

void init_motif()
{
	string temp = "";
	for (int i = 0; i < h_DataInp[0].size() - h_l; ++i)
	{
		temp = h_DataInp[0].substr(i, h_l);
		h_motif.push_back(temp);
	}
}

void File_Input()
{
	fi >> h_l >> h_d;
	fi.ignore();
	for (int i = 0; i<20; ++i)
	{
		getline(fi, h_DataInp[i]);
	}
}

int main(void)
{
	File_Input();
	init_motif();
	cout << h_motif.size() << endl;
	return 0;
}

